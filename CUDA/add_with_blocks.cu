
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 512

__global__ void add(int *a, int *b, int *c){
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main(void){
    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);
    
    a = 2;
    b = 7;

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    add<<<N,1>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}