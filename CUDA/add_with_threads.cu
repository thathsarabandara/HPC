
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 512

__global__ void add(int *a, int *b, int *c){
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main(void){
    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);
    
    a = 2;
    b = 7;

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    add<<<1,N>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}