
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(void){
    printf("Hello World from GPU!\n");
}

int main(void){
    mykernel<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
