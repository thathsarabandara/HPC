
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N (2048 * 2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}

int main(void){
    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);
    
    a = 2;
    b = 7;

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    add<<<N/THREADS_PER_BLOCK , THREADS_PER_BLOCK >>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}